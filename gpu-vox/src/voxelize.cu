#include "hip/hip_runtime.h"
#include "voxelize.cuh"
#include <math.h>

// CUDA Global Memory variables

// Debug counters for some sanity checks
#ifdef _DEBUG
__device__ size_t debug_d_n_voxels_marked = 0;
__device__ size_t debug_d_n_triangles = 0;
__device__ size_t debug_d_n_voxels_tested = 0;
#endif

// Morton LUTs for when we need them
__constant__ uint32_t morton256_x[256];
__constant__ uint32_t morton256_y[256];
__constant__ uint32_t morton256_z[256];

// Encode morton code using LUT table
int find_label(float d, float d1, float d2);

__device__ inline uint64_t mortonEncode_LUT(unsigned int x, unsigned int y, unsigned int z){
	uint64_t answer = 0;
	answer = morton256_z[(z >> 16) & 0xFF] |
		morton256_y[(y >> 16) & 0xFF] |
		morton256_x[(x >> 16) & 0xFF];
	answer = answer << 48 |
		morton256_z[(z >> 8) & 0xFF] |
		morton256_y[(y >> 8) & 0xFF] |
		morton256_x[(x >> 8) & 0xFF];
	answer = answer << 24 |
		morton256_z[(z)& 0xFF] |
		morton256_y[(y)& 0xFF] |
		morton256_x[(x)& 0xFF];
	return answer;
}

// Possible optimization: buffer bitsets (for now: Disabled because too much overhead)
//struct bufferedBitSetter{
//	unsigned int* voxel_table;
//	size_t current_int_location;
//	unsigned int current_mask;
//
//	__device__ __inline__ bufferedBitSetter(unsigned int* voxel_table, size_t index) :
//		voxel_table(voxel_table), current_mask(0) {
//		current_int_location = int(index / 32.0f);
//	}
//
//	__device__ __inline__ void setBit(size_t index){
//		size_t new_int_location = int(index / 32.0f);
//		if (current_int_location != new_int_location){
//			flush();
//			current_int_location = new_int_location;
//		}
//		unsigned int bit_pos = 31 - (unsigned int)(int(index) % 32);
//		current_mask = current_mask | (1 << bit_pos);
//	}
//
//	__device__ __inline__ void flush(){
//		if (current_mask != 0){
//			atomicOr(&(voxel_table[current_int_location]), current_mask);
//		}
//	}
//};

// Possible optimization: check bit before you set it - don't need to do atomic operation if it's already set to 1
// For now: overhead, so it seems
//__device__ __inline__ bool checkBit(unsigned int* voxel_table, size_t index){
//	size_t int_location = index / size_t(32);
//	unsigned int bit_pos = size_t(31) - (index % size_t(32)); // we count bit positions RtL, but array indices LtR
//	return ((voxel_table[int_location]) & (1 << bit_pos));
//}

// Set a bit in the giant voxel table. This involves doing an atomic operation on a 32-bit word in memory.
// Blocking other threads writing to it for a very short time
__device__ __inline__ void setBit(unsigned int* voxel_table, size_t index){
	size_t int_location = index / size_t(32);
	unsigned int bit_pos = size_t(31) - (index % size_t(32)); // we count bit positions RtL, but array indices LtR
	unsigned int mask = 1 << bit_pos;
	atomicOr(&(voxel_table[int_location]), mask);
}

__device__ __inline__
void find_voxel_label(float* a, float* b, float* c, float* label) {
    if (*a == *b || *a == *c) {
        *label = *a;
    }
    if (*b == *c){
        *label =  *b;
    }
    // In worst case, just use the max value from the three. Maybe we can replace it with the min value since unknown label is 100
    // however, that can stay as a future modification
    *label =  fmax(fmax(*a, *b), *c);
}


__device__ __inline__ void
setData(unsigned int *color_table, size_t index, glm::vec3 vec, glm::vec3 vec1, glm::vec3 vec2, glm::vec3 vec3) {
    size_t int_location = size_t (4 * index);
	color_table[int_location + 0] = (int) (255 * (vec.x + vec1.x + vec2.x )/3.0f);
    color_table[int_location + 1] = (int) (255 * (vec.y + vec1.y + vec2.y )/3.0f);
    color_table[int_location + 2] = (int) (255 * (vec.z + vec1.z + vec2.z )/3.0f);
    float label;
    find_voxel_label(&vec3.x, &vec3.y, &vec3.z, &label);
    color_table[int_location + 3] = (int) label;
}



// Main triangle voxelization method
__global__ void voxelize_triangle(voxinfo info, float* triangle_data, unsigned int* voxel_table,unsigned int* color_table, bool morton_order){
	size_t thread_id = threadIdx.x + blockIdx.x * blockDim.x;
	size_t stride = blockDim.x * gridDim.x;

	// Common variables used in the voxelization process
	glm::vec3 delta_p(info.unit.x, info.unit.y, info.unit.z);
	glm::vec3 grid_max(info.gridsize.x - 1, info.gridsize.y - 1, info.gridsize.z - 1); // grid max (grid runs from 0 to gridsize-1)

	while (thread_id < info.n_triangles){ // every thread works on specific triangles in its stride
        //		Since 9 more vertices added for the color info so we should skip 18
        // Another 3 added for making sure that we include label info
		size_t t = thread_id * 21; // triangle contains 9 vertices


		// COMPUTE COMMON TRIANGLE PROPERTIES
		// Move vertices to origin using bbox
		glm::vec3 v0 = glm::vec3(triangle_data[t], triangle_data[t + 1], triangle_data[t + 2]) - info.bbox.min;
		glm::vec3 v1 = glm::vec3(triangle_data[t + 3], triangle_data[t + 4], triangle_data[t + 5]) - info.bbox.min; 
		glm::vec3 v2 = glm::vec3(triangle_data[t + 6], triangle_data[t + 7], triangle_data[t + 8]) - info.bbox.min;

//		Color information
        glm::vec3 c0 = glm::vec3(triangle_data[t+9], triangle_data[t + 10], triangle_data[t + 11]);
        glm::vec3 c1 = glm::vec3(triangle_data[t+12], triangle_data[t + 13], triangle_data[t + 14]);
        glm::vec3 c2 = glm::vec3(triangle_data[t+15], triangle_data[t + 16], triangle_data[t + 17]);
        glm::vec3 ll = glm::vec3(triangle_data[t+18], triangle_data[t + 19], triangle_data[t + 20]);

		// Edge vectors
		glm::vec3 e0 = v1 - v0;
		glm::vec3 e1 = v2 - v1;
		glm::vec3 e2 = v0 - v2;
		// Normal vector pointing up from the triangle
		glm::vec3 n = glm::normalize(glm::cross(e0, e1));

		// COMPUTE TRIANGLE BBOX IN GRID
		// Triangle bounding box in world coordinates is min(v0,v1,v2) and max(v0,v1,v2)
		AABox<glm::vec3> t_bbox_world(glm::min(v0, glm::min(v1, v2)), glm::max(v0, glm::max(v1, v2)));
		// Triangle bounding box in voxel grid coordinates is the world bounding box divided by the grid unit vector
		AABox<glm::ivec3> t_bbox_grid;
		t_bbox_grid.min = glm::clamp(t_bbox_world.min / info.unit, glm::vec3(0.0f, 0.0f, 0.0f), grid_max);
		t_bbox_grid.max = glm::clamp(t_bbox_world.max / info.unit, glm::vec3(0.0f, 0.0f, 0.0f), grid_max);

		// PREPARE PLANE TEST PROPERTIES
		glm::vec3 c(0.0f, 0.0f, 0.0f);
		if (n.x > 0.0f) { c.x = info.unit.x; }
		if (n.y > 0.0f) { c.y = info.unit.y; }
		if (n.z > 0.0f) { c.z = info.unit.z; }
		float d1 = glm::dot(n, (c - v0));
		float d2 = glm::dot(n, ((delta_p - c) - v0));

		// PREPARE PROJECTION TEST PROPERTIES
		// XY plane
		glm::vec2 n_xy_e0(-1.0f*e0.y, e0.x);
		glm::vec2 n_xy_e1(-1.0f*e1.y, e1.x);
		glm::vec2 n_xy_e2(-1.0f*e2.y, e2.x);
		if (n.z < 0.0f) {
			n_xy_e0 = -n_xy_e0;
			n_xy_e1 = -n_xy_e1;
			n_xy_e2 = -n_xy_e2;
		}
		float d_xy_e0 = (-1.0f * glm::dot(n_xy_e0, glm::vec2(v0.x, v0.y))) + glm::max(0.0f, info.unit.x*n_xy_e0[0]) + glm::max(0.0f, info.unit.y*n_xy_e0[1]);
		float d_xy_e1 = (-1.0f * glm::dot(n_xy_e1, glm::vec2(v1.x, v1.y))) + glm::max(0.0f, info.unit.x*n_xy_e1[0]) + glm::max(0.0f, info.unit.y*n_xy_e1[1]);
		float d_xy_e2 = (-1.0f * glm::dot(n_xy_e2, glm::vec2(v2.x, v2.y))) + glm::max(0.0f, info.unit.x*n_xy_e2[0]) + glm::max(0.0f, info.unit.y*n_xy_e2[1]);
		// YZ plane
		glm::vec2 n_yz_e0(-1.0f*e0.z, e0.y);
		glm::vec2 n_yz_e1(-1.0f*e1.z, e1.y);
		glm::vec2 n_yz_e2(-1.0f*e2.z, e2.y);
		if (n.x < 0.0f) {
			n_yz_e0 = -n_yz_e0;
			n_yz_e1 = -n_yz_e1;
			n_yz_e2 = -n_yz_e2;
		}
		float d_yz_e0 = (-1.0f * glm::dot(n_yz_e0, glm::vec2(v0.y, v0.z))) + glm::max(0.0f, info.unit.y*n_yz_e0[0]) + glm::max(0.0f, info.unit.z*n_yz_e0[1]);
		float d_yz_e1 = (-1.0f * glm::dot(n_yz_e1, glm::vec2(v1.y, v1.z))) + glm::max(0.0f, info.unit.y*n_yz_e1[0]) + glm::max(0.0f, info.unit.z*n_yz_e1[1]);
		float d_yz_e2 = (-1.0f * glm::dot(n_yz_e2, glm::vec2(v2.y, v2.z))) + glm::max(0.0f, info.unit.y*n_yz_e2[0]) + glm::max(0.0f, info.unit.z*n_yz_e2[1]);
		// ZX plane
		glm::vec2 n_zx_e0(-1.0f*e0.x, e0.z);
		glm::vec2 n_zx_e1(-1.0f*e1.x, e1.z);
		glm::vec2 n_zx_e2(-1.0f*e2.x, e2.z);
		if (n.y < 0.0f) {
			n_zx_e0 = -n_zx_e0;
			n_zx_e1 = -n_zx_e1;
			n_zx_e2 = -n_zx_e2;
		}
		float d_xz_e0 = (-1.0f * glm::dot(n_zx_e0, glm::vec2(v0.z, v0.x))) + glm::max(0.0f, info.unit.x*n_zx_e0[0]) + glm::max(0.0f, info.unit.z*n_zx_e0[1]);
		float d_xz_e1 = (-1.0f * glm::dot(n_zx_e1, glm::vec2(v1.z, v1.x))) + glm::max(0.0f, info.unit.x*n_zx_e1[0]) + glm::max(0.0f, info.unit.z*n_zx_e1[1]);
		float d_xz_e2 = (-1.0f * glm::dot(n_zx_e2, glm::vec2(v2.z, v2.x))) + glm::max(0.0f, info.unit.x*n_zx_e2[0]) + glm::max(0.0f, info.unit.z*n_zx_e2[1]);



		// test possible grid boxes for overlap
		for (int z = t_bbox_grid.min.z; z <= t_bbox_grid.max.z; z++){
			for (int y = t_bbox_grid.min.y; y <= t_bbox_grid.max.y; y++){
				for (int x = t_bbox_grid.min.x; x <= t_bbox_grid.max.x; x++){
					// size_t location = x + (y*info.gridsize) + (z*info.gridsize*info.gridsize);
					// if (checkBit(voxel_table, location)){ continue; }
#ifdef _DEBUG
					atomicAdd(&debug_d_n_voxels_tested, 1);
#endif
					// TRIANGLE PLANE THROUGH BOX TEST
					glm::vec3 p(x*info.unit.x, y*info.unit.y, z*info.unit.z);
					float nDOTp = glm::dot(n, p);
					if ((nDOTp + d1) * (nDOTp + d2) > 0.0f) { continue; }

					// PROJECTION TESTS
					// XY
					glm::vec2 p_xy(p.x, p.y);
					if ((glm::dot(n_xy_e0, p_xy) + d_xy_e0) < 0.0f){ continue; }
					if ((glm::dot(n_xy_e1, p_xy) + d_xy_e1) < 0.0f){ continue; }
					if ((glm::dot(n_xy_e2, p_xy) + d_xy_e2) < 0.0f){ continue; }

					// YZ
					glm::vec2 p_yz(p.y, p.z);
					if ((glm::dot(n_yz_e0, p_yz) + d_yz_e0) < 0.0f){ continue; }
					if ((glm::dot(n_yz_e1, p_yz) + d_yz_e1) < 0.0f){ continue; }
					if ((glm::dot(n_yz_e2, p_yz) + d_yz_e2) < 0.0f){ continue; }

					// XZ	
					glm::vec2 p_zx(p.z, p.x);
					if ((glm::dot(n_zx_e0, p_zx) + d_xz_e0) < 0.0f){ continue; }
					if ((glm::dot(n_zx_e1, p_zx) + d_xz_e1) < 0.0f){ continue; }
					if ((glm::dot(n_zx_e2, p_zx) + d_xz_e2) < 0.0f){ continue; }

#ifdef _DEBUG
					atomicAdd(&debug_d_n_voxels_marked, 1);
#endif

					if (morton_order){
						size_t location = mortonEncode_LUT(x, y, z);
						setBit(voxel_table, location);
                        setData(color_table, location, c0, c1, c2, ll);
					} else {
						size_t location = static_cast<size_t>(x) + (static_cast<size_t>(y)* static_cast<size_t>(info.gridsize.x)) + (static_cast<size_t>(z)* static_cast<size_t>(info.gridsize.y)* static_cast<size_t>(info.gridsize.x));
						setBit(voxel_table, location);
                        setData(color_table, location, c0, c1, c2, ll);
					}
					continue;
				}
			}
		}
#ifdef _DEBUG
		atomicAdd(&debug_d_n_triangles, 1);
#endif
        //		Since color info was added, just skip those here
        thread_id = thread_id + stride;
	}
}

void voxelize(const voxinfo& v, float* triangle_data, unsigned int* vtable, unsigned int* colortable, bool useThrustPath, bool morton_code) {
	float   elapsedTime;

	// These are only used when we're not using UNIFIED memory
	unsigned int* dev_vtable; // DEVICE pointer to voxel_data
	unsigned int* dev_colortable; // DEVICE pointer to voxel_data
	size_t vtable_size; // vtable size


	// Create timers, set start time
	hipEvent_t start_vox, stop_vox;
	checkCudaErrors(hipEventCreate(&start_vox));
	checkCudaErrors(hipEventCreate(&stop_vox));

	// Copy morton LUT if we're encoding to morton
	if (morton_code){
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(morton256_x), host_morton256_x, 256 * sizeof(uint32_t)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(morton256_y), host_morton256_y, 256 * sizeof(uint32_t)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(morton256_z), host_morton256_z, 256 * sizeof(uint32_t)));
	}

	// Estimate best block and grid size using CUDA Occupancy Calculator
	int blockSize;   // The launch configurator returned block size 
	int minGridSize; // The minimum grid size needed to achieve the  maximum occupancy for a full device launch 
	int gridSize;    // The actual grid size needed, based on input size 
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, voxelize_triangle, 0, 0);
	// Round up according to array size 
	gridSize = (v.n_triangles + blockSize - 1) / blockSize;

	if (useThrustPath) { // We're not using UNIFIED memory
		vtable_size = ((size_t)v.gridsize.x * v.gridsize.y * v.gridsize.z) / (size_t) 8.0;
		fprintf(stdout, "[Voxel Grid] Allocating %llu kB of DEVICE memory for Voxel Grid\n", size_t(vtable_size / 1024.0f));
		checkCudaErrors(hipMalloc(&dev_vtable, vtable_size));
		checkCudaErrors(hipMemset(dev_vtable, 0, vtable_size));
//		Do the same for colors
        fprintf(stdout, "[Color Grid] Allocating %llu kB of DEVICE memory for Color Grid\n", size_t((vtable_size) * size_t(4)* (size_t) 32/ 1024.0f));
        checkCudaErrors(hipMalloc(&dev_colortable, vtable_size * size_t(4) * (size_t) 32));
        checkCudaErrors(hipMemset(dev_colortable, 0, vtable_size * size_t(4) * (size_t) 32 ));
		// Start voxelization
		checkCudaErrors(hipEventRecord(start_vox, 0));
		voxelize_triangle <<<gridSize, blockSize >>> (v, triangle_data, dev_vtable, dev_colortable, morton_code);
	}
	else { // UNIFIED MEMORY 
		checkCudaErrors(hipEventRecord(start_vox, 0));
		voxelize_triangle << <gridSize, blockSize >> > (v, triangle_data, vtable, colortable, morton_code);
	}

	hipDeviceSynchronize();
	checkCudaErrors(hipEventRecord(stop_vox, 0));
	checkCudaErrors(hipEventSynchronize(stop_vox));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start_vox, stop_vox));
	printf("[Perf] Voxelization GPU time: %.1f ms\n", elapsedTime);

	// If we're not using UNIFIED memory, copy the voxel table back and free all
	if (useThrustPath){
		fprintf(stdout, "[Voxel Grid] Copying %llu kB to page-locked HOST memory\n", size_t(vtable_size / 1024.0f));
		checkCudaErrors(hipMemcpy((void*)vtable, dev_vtable, vtable_size, hipMemcpyDefault));
		fprintf(stdout, "[Voxel Grid] Freeing %llu kB of DEVICE memory\n", size_t(vtable_size / 1024.0f));
//		Same for the colors
        fprintf(stdout, "[Color Grid] Copying %llu kB to page-locked HOST memory\n", size_t(vtable_size * size_t(4) * (size_t) 32 / 1024.0f));
        checkCudaErrors(hipMemcpy((void*)colortable, dev_colortable, vtable_size * size_t(4) * size_t (32), hipMemcpyDefault));
        fprintf(stdout, "[Color Grid] Freeing %llu kB of DEVICE memory\n", size_t(vtable_size * size_t(4) * (size_t) 32 / 1024.0f));
		checkCudaErrors(hipFree(dev_vtable));
		checkCudaErrors(hipFree(dev_colortable));
	}

	// SANITY CHECKS
#ifdef _DEBUG
	size_t debug_n_triangles, debug_n_voxels_marked, debug_n_voxels_tested;
	checkCudaErrors(hipMemcpyFromSymbol((void*)&(debug_n_triangles),debug_d_n_triangles, sizeof(debug_d_n_triangles), 0, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpyFromSymbol((void*)&(debug_n_voxels_marked), debug_d_n_voxels_marked, sizeof(debug_d_n_voxels_marked), 0, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpyFromSymbol((void*) & (debug_n_voxels_tested), debug_d_n_voxels_tested, sizeof(debug_d_n_voxels_tested), 0, hipMemcpyDeviceToHost));
	printf("[Debug] Processed %llu triangles on the GPU \n", debug_n_triangles);
	printf("[Debug] Tested %llu voxels for overlap on GPU \n", debug_n_voxels_tested);
	printf("[Debug] Marked %llu voxels as filled (includes duplicates!) \n", debug_n_voxels_marked);
#endif

	// Destroy timers
	checkCudaErrors(hipEventDestroy(start_vox));
	checkCudaErrors(hipEventDestroy(stop_vox));
}
